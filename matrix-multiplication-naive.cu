#include "hip/hip_runtime.h"
#include "solve.h"
#include <hip/hip_runtime.h>

__global__ void matrix_multiplication_kernel(const float* A, const float* B, float* C, int M, int K, int N) {
    const uint x = blockIdx.x * blockDim.x + threadIdx.x;
    const uint y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < M && y < N) {
        float tmp = 0.0;
        for (int i = 0; i < K; ++i) {
            tmp += A[x * K + i] * B[i * N + y];
        }
        C[x * N + y] = tmp;
    }
}

// A, B, C are device pointers (i.e. pointers to memory on the GPU)
void solve(const float* A, const float* B, float* C, int M, int K, int N) {
    dim3 threadsPerBlock(32, 32);
    dim3 blocksPerGrid((M + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (N + threadsPerBlock.y - 1) / threadsPerBlock.y);
    
    matrix_multiplication_kernel<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, M, K, N);
    hipDeviceSynchronize();
}
