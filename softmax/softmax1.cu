#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <stdio.h>
#include <c10/cuda/CUDAException.h>
#include <cfloat>

#define CHECK_CUDA(x) TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

inline unsigned int cdiv(unsigned int a, unsigned int b) { return (a + b - 1) / b;}

__global__ void softmax1_kernel(const float* x, float* y, int rows, int cols) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < rows) {
        float max_val = -FLT_MAX;
        for (int col = 0; col < cols; col++) {
            int i = row * cols + col;
            max_val = max(max_val, x[i]);
        }
        float sum = 0;
        for (int col = 0; col < cols; col++) {
            int i = row * cols + col;
            sum += exp(x[i] - max_val);
        }
        for (int col = 0; col < cols; col++) {
            int i = row * cols + col;
            y[i] = exp(x[i] - max_val) / sum;
        }
    }
}

torch::Tensor softmax1(const torch::Tensor& x) {
    auto x_flatten = x.reshape({-1, x.size(-1)});
    auto y = torch::empty_like(x_flatten);
    dim3 block(1024);
    dim3 grid(cdiv(x_flatten.size(0), block.x));
    softmax1_kernel<<<grid, block>>>(x_flatten.data_ptr<float>(), y.data_ptr<float>(), x_flatten.size(0), x_flatten.size(1));
    return y.view_as(x);
}
