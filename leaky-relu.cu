#include "hip/hip_runtime.h"
#include "solve.h"
#include <hip/hip_runtime.h>

__global__ void leaky_relu_kernel(const float* input, float* output, int N) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < N) {
        float x = input[i];
        output[i] = fmaxf(x, 0.f) + 0.01f * fminf(x, 0.f);
    }
}

// input, output are device pointers (i.e. pointers to memory on the GPU)
void solve(const float* input, float* output, int N) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    
    leaky_relu_kernel<<<blocksPerGrid, threadsPerBlock>>>(input, output, N);
    hipDeviceSynchronize();
}
