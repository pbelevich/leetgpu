#include "hip/hip_runtime.h"
#include "solve.h"
#include <hip/hip_runtime.h>

inline __device__ __host__ unsigned int cdiv(unsigned int a, unsigned int b) { return (a + b - 1) / b;}

#define WARP_SIZE 32
#define THREADS_PER_BLOCK 256
#define STRIDE_FACTOR 8
#define BLOCK_SIZE STRIDE_FACTOR*THREADS_PER_BLOCK

__device__ void warp_reduce(volatile float* smem, unsigned int tid){
    #pragma unroll
    for (int s = WARP_SIZE; s > 0; s >>= 1) {
        smem[tid] += smem[tid + s];
    }
}

__global__ void reduction_kernel(const float* input, float* output, int N) {
    __shared__ float smem[THREADS_PER_BLOCK];
    auto tid = threadIdx.x;

    auto block_start = blockIdx.x * BLOCK_SIZE;
    auto idx = block_start + tid;
    float sum = idx < N ? input[idx] : 0.0f;
    for(int i = 1; i < STRIDE_FACTOR; ++i){
        auto idx = block_start + i * THREADS_PER_BLOCK + tid;
        if (idx < N){
            sum += input[idx];
        }
    }

    // SMEM reduction
    smem[tid] = sum;
    __syncthreads();
    for (int s = THREADS_PER_BLOCK >> 1; s > WARP_SIZE; s >>= 1) {
        if (tid < s) {
            smem[tid] += smem[tid + s];
        }
        __syncthreads();
    }

    // Warp reduction
    if (tid < WARP_SIZE) {
        warp_reduce(smem, tid);
    }

    // Grid reduction
    if (tid == 0) {
        atomicAdd(output, smem[0]);
    }
}

// input, output are device pointers
void solve(const float* input, float* output, int N) {
    dim3 threads(THREADS_PER_BLOCK);
    dim3 blocks(cdiv(N, THREADS_PER_BLOCK * STRIDE_FACTOR));
    reduction_kernel<<<blocks, threads>>>(input, output, N);
}
