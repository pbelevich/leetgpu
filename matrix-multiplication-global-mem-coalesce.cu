#include "hip/hip_runtime.h"
#include "solve.h"
#include <hip/hip_runtime.h>

#define CEIL_DIV(M, N) (((M) + (N)-1) / (N))
#define BLOCKSIZE 32

__global__ void matrix_multiplication_kernel(const float* A, const float* B, float* C, int M, int K, int N) {
    const int cRow = blockIdx.x * BLOCKSIZE + (threadIdx.x / BLOCKSIZE);
    const int cCol = blockIdx.y * BLOCKSIZE + (threadIdx.x % BLOCKSIZE);

    // if statement is necessary to make things work under tile quantization
    if (cRow < M && cCol < N) {
        float tmp = 0.0;
        for (int i = 0; i < K; ++i) {
            tmp += A[cRow * K + i] * B[i * N + cCol];
        }
        C[cRow * N + cCol] = tmp;
    }
}

// A, B, C are device pointers (i.e. pointers to memory on the GPU)
void solve(const float* A, const float* B, float* C, int M, int K, int N) {
    dim3 blocksPerGrid(CEIL_DIV(M, 32), CEIL_DIV(N, 32));
    dim3 threadsPerBlock(32 * 32);
    matrix_multiplication_kernel<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, M, K, N);
    hipDeviceSynchronize();
}
