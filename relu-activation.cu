#include "hip/hip_runtime.h"
#include "solve.h"
#include <hip/hip_runtime.h>

__global__ void relu_kernel(const float* input, float* output, int N) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < N) {
        output[i] = max(0.0, input[i]);
    }
}

// input, output are device pointers (i.e. pointers to memory on the GPU)
void solve(const float* input, float* output, int N) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    relu_kernel<<<blocksPerGrid, threadsPerBlock>>>(input, output, N);
    hipDeviceSynchronize();
}
