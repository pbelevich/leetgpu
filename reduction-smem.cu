#include "hip/hip_runtime.h"
#include "solve.h"
#include <hip/hip_runtime.h>

inline __device__ __host__ unsigned int cdiv(unsigned int a, unsigned int b) { return (a + b - 1) / b;}

#define WARP_SIZE 32

__global__ void reduction_kernel(const float* input, float* output, int N, int stride_factor) {
    extern __shared__ float smem[];
    auto tid = threadIdx.x;
    auto thread_per_block = blockDim.x;
    auto block_size = thread_per_block * stride_factor;
    auto block_start = blockIdx.x * block_size;
    
    // Stride reduction
    auto sum = 0.0;
    for (int i = 0; i < stride_factor; ++i) {
        auto idx = block_start + i * thread_per_block + tid;
        if (idx < N) {
            sum += input[idx];
        }
    }

    // SMEM reduction
    smem[tid] = sum;
    __syncthreads();
    for (int s = thread_per_block >> 1; s > 0; s >>= 1) {
        if (tid < s) {
            smem[tid] += smem[tid + s];
        }
        __syncthreads();
    }

    // Grid reduction
    if (tid == 0) {
        atomicAdd(output, smem[0]);
    }
}

// input, output are device pointers
void solve(const float* input, float* output, int N) {
    auto threads_per_block = 256;
    auto stride_factor = 16;
    dim3 threads(threads_per_block);
    dim3 blocks(cdiv(N, threads_per_block * stride_factor));
    auto smem_size = threads_per_block * sizeof(float);
    reduction_kernel<<<blocks, threads, smem_size>>>(input, output, N, stride_factor);
}
